/**
 * @file check_compute_capability.cpp
 * @author qiujiandong <1335521934@qq.com>
 * @date 2024-04-16
 * @brief
 *
 *
 */


#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  std::cout << "Compute Capability: " << prop.major << prop.minor << std::endl;
  return 0;
}
